#include "hip/hip_runtime.h"
#include "Array2D_CUDA.h"
#include "Array2D.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#define BLOCK_SIZE 1024

template <class T>
__global__ void pow2(T* in, T* out, size_t N){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (idx < N)out[idx] = in[idx] * in[idx];
}

template <class T>
void ArrayPow2_CUDA(Array2D<T>& in, Array2D<T>& result) {
    Array2D< Cutype<T> > in_d(in);
    size_t N = in.size();
    pow2 <<< (N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE >>> (in_d.begin(), in_d.begin(), in.size());
    //double* in2 = in_d.begin();
    //double* out2 = in2;
    //pow2 <<< (N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE >>> (in2, out2, in.size());
    T* arr_h = in.begin();
    T* arr_d = in_d.begin();
    hipMemcpy(arr_h, arr_d, sizeof(T) * N, hipMemcpyDeviceToHost);
}

template void ArrayPow2_CUDA(Array2D<float>&, Array2D<float>&);
template __global__ void pow2(float*, float*, size_t);

/*
__global__ void pow2(double* in, double* out, size_t N){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if (idx < N)out[idx] = in[idx] * in[idx];
}

void ArrayPow2_CUDA(Array2D<double>& in, Array2D<double>& result) {
    Array2D< Cutype<double> > in_d(in);
    size_t N = in.size();
    double* arr_h = in.begin();
    double* arr_d = in_d.begin();
    pow2 <<< (N - 1) / BLOCK_SIZE + 1, BLOCK_SIZE >>> (arr_h, arr_h, N);
    hipMemcpy(arr_h, arr_d, sizeof(double) * N, hipMemcpyDeviceToHost);
}
*/
